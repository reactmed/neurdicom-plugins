#include "hip/hip_runtime.h"
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__

// This is slightly mental, but gets it to properly index device function calls like __popc and whatever.
#define __HIPCC__

#include <hip/device_functions.h>

// These headers are all implicitly present when you compile CUDA with clang. Clion doesn't know that, so
// we include them explicitly to make the indexer happy. Doing this when you actually build is, obviously,
// a terrible idea :D
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>

#endif // __JETBRAINS_IDE__

#define BLOCKDIM 1024

struct Cluster {
    float sum;
    int count;
};

__device__ Cluster clusters_d[(N + BLOCKDIM - 1) / BLOCKDIM];

__device__ float euclidianDist(const float a, const float b) {
    float dist = a - b;
    return hypotf(dist, dist);
}

__global__ void relabel(const float *src, const float *clusters, int n, int nClusters, int *labels) {
    int pos = threadIdx.x + blockIdx.x * blockDim.x;
    if (pos < n) {
        float minDist = 1.0f;
        int clusterIndex = 0;
        for (int c = 0; c < nClusters; c++) {
            float dist = euclidianDist(src[pos], clusters[c]);
            if (dist <= minDist) {
                clusterIndex = c;
                minDist = dist;
            }
        }
        labels[pos] = clusterIndex;
    }
}

__global__ void calculateClusters(const float *src, const int *labels, int n, int clusterIndex) {
    extern __shared__ Cluster
    _clusters[];
    int pos = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;
    _clusters[tid] = Cluster();
    _clusters[tid].sum = 0.0f;
    _clusters[tid].count = 0;
    if (pos < n && labels[pos] == clusterIndex) {
        _clusters[tid].sum = src[pos];
        _clusters[tid].count = 1;
    }
    __syncthreads();
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            _clusters[tid].sum += _clusters[tid + stride].sum;
        }
        __syncthreads();
        if (threadIdx.x < stride) {
            _clusters[tid].count += _clusters[tid + stride].count;
        }
        __syncthreads();
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        clusters_d[blockIdx.x].sum = _clusters[0].sum;
        clusters_d[blockIdx.x].count = _clusters[0].count;
    }
}

__global__ void findCenters(int n, int clusterIndex, float *dst) {
    extern __shared__ Cluster
    _clusters[];
    int pos = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;
    _clusters[tid] = clusters_d[pos];
    __syncthreads();
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            _clusters[tid].sum += _clusters[tid + stride].sum;
        }
        __syncthreads();
        if (tid < stride) {
            _clusters[tid].count += _clusters[tid + stride].count;
        }
        __syncthreads();
    }
    __syncthreads();
    if (tid == 0) {
        dst[clusterIndex] = _clusters[0].count > 0 ? _clusters[0].sum / (_clusters[0].count * 1.0f) : 0.0f;
    }
}